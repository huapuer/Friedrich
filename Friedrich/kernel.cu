#include "hip/hip_runtime.h"
/*
TODO: ������������ʼֵ
TODO: HEBBȨ����ǿ

DESG: �����߼��㣬�߼�������������һ���ֻ�ȫ������
	  �������������Լ����߼�������ã�����㱻����ʱ����ͬ�������߼���״̬��������߼��������������񵽵�����
DESG: Host Scheduler��Slave Batch���н��ʵ��Host��Slave������ҵ������Host�˵��ȿ���

TODO: ����������������߼���֮���״̬ͬ���߼�(scheduling debug), ������ɣ��������߼�������������(layer1->layer2)
TODO: ���Ӳ�ͬ���ӷ�ʽ��1:1/n:n��
TODO: ���Ӳ�����Ȩ������֧�֣�mute_fnΪNULL��
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "mpi.h" 
#include <Windows.h>
#include <memory.h>

#include "../../Ludwig/Ludwig/ludwig_neural_network.h"
#include "../../Ludwig/Ludwig/ludwig_net.h"
#pragma comment(lib, "../../Ludwig/x64/Debug/Ludwig.lib")

#include "net.h"

#define ERROR(format,...) do{fprintf(stderr,format,##__VA_ARGS__);system("pause");exit(1);}while(0)
#define DEBUG

enum execute_type {
	EXECUTE_LAYER,
	EXECUTE_LINK
};

typedef void(*fp_integrate)(gen_t*, int);
typedef void(*fp_mute)(gen_w*, const unsigned long long);
typedef void(*fp_clear_push)(const gen_t *, gen_t *, int, gen_w*, const int, const unsigned long long);
typedef void(*fp_push)(const gen_t*, gen_t*, int, gen_w*, const int, const unsigned long long);

struct executable {
	unsigned long long gen;
	execute_type type;
	layer_t* s;
	link* l;
	layer_t* t;
	executable* pre;
	executable* next;
	bool done;
};

struct map {
	int size;
	float* dev_t;
};

__constant__ map w_mutes;
int thread_num;
layer_t* layer_list = 0;

__global__ void default_integrate(gen_t* s, int offset) {
	int i = threadIdx.x + offset;
	if (s[i].t > 3.0) {
		s[i].t = 1.0;
	}else {
		s[i].t = 0.0;
	}
}

__global__ void default_mute(gen_w* w, const unsigned long long gen) {
	int i = threadIdx.x;
	if (w[i].working_gen == gen) {
		int gap = gen - w[i].gen;
		if (gap == 1) {
			w[i].stage++;
		}
		else {
			w[i].stage = 0;
		}
		if (w[i].stage < w_mutes.size) {
			w[i].t += w_mutes.dev_t[w->stage];
		}
		w[i].gen = gen;
	}
}

__global__ void default_clear_push_full(const gen_t *s, gen_t *t, int soffset, int toffset, gen_w* w, const int ts, const unsigned long long gen)
{
	int j = threadIdx.x + soffset;
	for (int i = toffset; i < ts; i++) {
		t[i].t = 0.0;
		if (s[j].t > 0.0) {
			t[i].t += s[j].t * w[i*j + j].t;
			w[i*j + j].working_gen = gen;
		}
	}
}

__global__ void default_push_full(const gen_t *s, gen_t *t, int soffset, int toffset, gen_w* w, const int ts, const unsigned long long gen)
{
	int j = threadIdx.x + soffset;
	for (int i = toffset; i < ts; i++) {
		if (s[j].t > 0.0) {
			t[i].t += s[j].t * w[i*j + j].t;
			w[i*j + j].working_gen = gen;
		}
	}
}

__global__ void default_clear_pull_forward(const gen_t *s, gen_t *t, int soffset, int toffset, gen_w* w, const int ss, const unsigned long long gen)
{
	int i = threadIdx.x + toffset;
	int j = soffset;
	t[i].t = 0.0;
	if (s[j].t > 0.0) {
		t[i].t += s[j].t * w[j].t;
		w[j].working_gen = gen;
	}
}

__global__ void default_pull_forward(const gen_t *s, gen_t *t, int soffset, int toffset, gen_w* w, const int ss, const unsigned long long gen)
{
	int i = threadIdx.x + toffset;
	int j = soffset;
	t[i].t = 0.0;
	if (s[j].t > 0.0) {
		t[i].t += s[j].t * w[j].t;
		w[j].working_gen = gen;
	}
}

__global__ void default_clear_pull_full(const gen_t *s, gen_t *t, int soffset, int toffset, gen_w* w, const int ss, const unsigned long long gen)
{
	int i = threadIdx.x + toffset;
	for (int j = soffset; j < ss; j++) {
		t[i].t = 0.0;
		if (s[j].t > 0.0) {
			t[i].t += s[j].t * w[i*j + j].t;
			w[i*j + j].working_gen = gen;
		}
	}
}

__global__ void default_pull_full(const gen_t *s, gen_t *t, int soffset, int toffset, gen_w* w, const int ss, const unsigned long long gen)
{
	int i = threadIdx.x + toffset;
	for (int j = soffset; j < ss; j++) {
		if (s[j].t > 0.0) {
			t[i].t += s[j].t * w[i*j + j].t;
			w[i*j + j].working_gen = gen;
		}
	}
}


void default_init_device() {
	layer_t* next = pick_layer(0);
	while (next) {
		int size = next->size;
		if (size > 0) {
			next->t = (gen_t*)malloc(sizeof(gen_t)*size);
			//TODO: initialize gen_t?
			hipMalloc((void**)&next->dev_t[0], size * sizeof(gen_t));
			hipMemcpy(next->dev_t[0], next->t, size * sizeof(gen_t), hipMemcpyHostToDevice);
			hipMalloc((void**)&next->dev_t[1], size * sizeof(gen_t));
			hipMemcpy(next->dev_t[1], next->t, size * sizeof(gen_t), hipMemcpyHostToDevice);
		}
		next = next->follow;
	}
}

executable* new_executable(int gen, execute_type type, layer_t* s, link* l, layer_t* t) {
	executable* ret = (executable*)malloc(sizeof(executable));
	memset(ret, 0, sizeof(executable));
	ret->gen = gen;
	ret->type = type;
	ret->s = s;
	ret->l = l;
	ret->t = t;
	ret->done = false;
	return ret;
}

void append_executable(executable** head, executable** tail, executable* n) {
	if (!*head) {
		*head = n;
	}
	if (*tail) {
		(*tail)->next = n;
		(*tail)->next->pre = *tail;
		(*tail) = (*tail)->next;
	}
	else {
		*tail = n;
	}
}

void remove_executable(executable** head, executable** tail, executable* e) {
	if (e->pre) {
		e->pre->next = e->next;
		if (!e->pre->pre) {
			*head = e->pre;
			if (*head) {
				(*head)->next = e->next;
			}
		}
	}
	else {
		*head = e->next;
		if (*head) {
			(*head)->next = e->next->next;
		}
	}
	if (e->next) {
		e->next->pre = e->pre;
		if (!e->next->next) {
			*tail = e->next;
			if (*tail) {
				(*tail)->pre = e->pre;
			}
		}
	}
	else {
		*tail = e->pre;
		if (*tail) {
			(*tail)->pre = e->pre->pre;
		}
	}
}

void swap_layer_dev(layer_t* l) {
	int tmp;
	tmp = l->cur_s_dev_t;
	l->cur_s_dev_t = l->cur_t_dev_t;
	l->cur_t_dev_t = tmp;
}

void wrap_layers(executable* task, layer_t** s_phisical, layer_t** t_phisical, layer_t** s_logical, layer_t** t_logical) {
	switch (task->s->type) {
	case LAYER_PHSICAL:
		*s_phisical = task->s;
		*s_logical = *s_phisical;
		break;
	case LAYER_LOGICAL:
		*s_phisical = task->s->phsical;
		switch (task->s->delegate) {
		case true:
			*s_logical = *s_phisical;
			break;
		case false:
			*s_logical = task->s;
			break;
		}
		break;
	}
	switch (task->t->type) {
	case LAYER_PHSICAL:
		*t_phisical = task->t;
		*t_logical = *t_phisical;
		break;
	case LAYER_LOGICAL:
		*t_phisical = task->t->phsical;
		switch (task->t->delegate) {
		case true:
			*t_logical = *t_phisical;
			break;
		case false:
			*t_logical = task->t;
			break;
		}
		break;
	}
}

void execute(executable* head, int max_gen) {
	unsigned long long gen = 1;
	unsigned long long batch = 0;
	const int task_width = 10;
	int tasks = 0;
	hipStream_t streams[task_width];
	for (int i = 0; i<task_width; i++)
	{
		hipStreamCreate(&streams[i]);
	}

	executable* link_task_head = NULL;
	executable* link_task_tail = NULL;
	executable* layer_task_head = head;
	executable* layer_task_tail = head;

	while (layer_task_head) {
		//critical region begin
		//excute
		
		batch++;
		tasks = 0;

		if (layer_task_head->gen > gen) {
#ifdef DEBUG
			if (gen >= max_gen) {
				break;
			}
#endif
			gen++;

			executable* link_task = link_task_head;
			while (link_task) {			
				while (link_task && tasks < task_width) {
					default_mute << <link_task->l->size / thread_num + 1, link_task->l->size>thread_num ? thread_num : link_task->l->size, 0, streams[tasks] >> >(link_task->l->dev_t, gen);
					remove_executable(&link_task_head, &link_task_tail, link_task);
					tasks++;
					link_task->l->mutating_batch = batch;
#ifdef DEBUG
					layer_t *s_phisical, *t_phisical, *s_logical, *t_logical;
					wrap_layers(link_task, &s_phisical, &t_phisical, &s_logical, &t_logical);
					fprintf(stdout, "GEN:%d BATCH:%d JOB:%s FROM:%d TO:%d\n", gen, batch, "MUTW", s_logical->id, t_logical->id);
#endif
					executable* tmp = link_task;
					link_task = link_task->next;
					free(tmp);
				}
				if (tasks == task_width) {
					for (int i = 0; i < tasks; i++) {
						hipStreamSynchronize(streams[i]);
					}
					batch++;
					tasks = 0;
				}
			}

			pick_layer(0)->integrated_gen = gen;
			layer_task_head->pre=new_executable(gen, EXECUTE_LAYER, pick_layer(0), pick_layer(0)->next, pick_layer(0)->next->layer);
			layer_task_head->pre->next = layer_task_head;
			layer_task_head = layer_task_head->pre;
		}

		executable* layer_task = layer_task_head;
		while (layer_task && tasks < task_width) {
			if (layer_task->gen == gen && tasks < task_width) {
				layer_t *s_phisical, *t_phisical, *s_logical, *t_logical;
				wrap_layers(layer_task, &s_phisical, &t_phisical, &s_logical, &t_logical);
				if (s_phisical->swap_gen != gen) {
					swap_layer_dev(s_phisical);
					s_phisical->swap_gen = gen;
				}
				if (t_phisical->swap_gen != gen) {
					swap_layer_dev(t_phisical);
					t_phisical->swap_gen = gen;
				}
				if (s_logical->integrated_gen != gen) {
					if (s_phisical->integrated_gen != gen) {
						default_integrate << <s_logical->size / thread_num + 1, s_logical->size>thread_num ? thread_num : s_logical->size, 0, streams[tasks] >> > (s_phisical->dev_t[s_phisical->cur_s_dev_t], s_logical->offset);
						tasks++;
						s_phisical->integrated_gen = gen;
					}
					s_logical->integrated_gen = gen;
					s_logical->integrating_batch = batch;
#ifdef DEBUG
					fprintf(stdout, "GEN:%d BATCH:%d JOB:%s FROM:%d TO:%d BUFF:%d\n", gen, batch, "INTE", s_logical->id, t_logical->id, s_phisical->cur_s_dev_t);
#endif
				}
				else if(s_logical->integrating_batch != batch){
					if (layer_task->l->mutating_batch != batch) {
						if (t_logical->working_gen != gen) {
							switch (layer_task->l->type) {
							case LINK_FORWARD:
								default_clear_pull_forward << <t_logical->size / thread_num + 1, t_logical->size>thread_num ? thread_num : t_logical->size, 0, streams[tasks] >> > (s_phisical->dev_t[s_phisical->cur_s_dev_t], t_phisical->dev_t[t_phisical->cur_t_dev_t], s_logical->offset, t_logical->offset, layer_task->l->dev_t, s_logical->size, gen);
								break;
							case LINK_FULL:
								if (s_logical->size <= t_logical->size) {
									default_clear_pull_full << <t_logical->size / thread_num + 1, t_logical->size>thread_num ? thread_num : t_logical->size, 0, streams[tasks] >> > (s_phisical->dev_t[s_phisical->cur_s_dev_t], t_phisical->dev_t[t_phisical->cur_t_dev_t], s_logical->offset, t_logical->offset, layer_task->l->dev_t, s_logical->size, gen);
								}
								else {
									default_clear_push_full << <s_logical->size / thread_num + 1, s_logical->size>thread_num ? thread_num : s_logical->size, 0, streams[tasks] >> > (s_phisical->dev_t[s_phisical->cur_s_dev_t], t_phisical->dev_t[t_phisical->cur_t_dev_t], s_logical->offset, t_logical->offset, layer_task->l->dev_t, t_logical->size, gen);
								}
							}
							tasks++;
							remove_executable(&layer_task_head, &layer_task_tail, layer_task);
							layer_task->done = true;
							link* next_link = t_logical->next;
							while (next_link) {
								executable* n = new_executable(gen + 1, EXECUTE_LAYER, t_logical, next_link, next_link->layer);
								append_executable(&layer_task_head, &layer_task_tail, n);
								next_link = next_link->another;
							}
							if (t_logical != t_phisical) {
								if (t_phisical->working_gen != gen) {
									link* next_link = t_phisical->next;
									while (next_link) {
										executable* n = new_executable(gen + 1, EXECUTE_LAYER, t_phisical, next_link, next_link->layer);
										append_executable(&layer_task_head, &layer_task_tail, n);
										next_link = next_link->another;
									}
								}
							}
							else {
								layer_t* next = t_logical->logical_head;
								while (next) {
									if (!next->delegate) {
										link* next_link = next->next;
										while (next_link) {
											executable* n = new_executable(gen + 1, EXECUTE_LAYER, next, next_link, next_link->layer);
											append_executable(&layer_task_head, &layer_task_tail, n);
											next_link = next_link->another;
										}
									}
									next->working_gen = gen;
									next->working_batch = batch;
									next = next->next_logical;
								}
							}
							t_logical->working_gen = gen;
							t_logical->working_batch = batch;
							t_phisical->working_gen = gen;
							t_phisical->working_batch = batch;
#ifdef DEBUG
							fprintf(stdout, "GEN:%d BATCH:%d JOB:%s FROM:%d TO:%d BUFF:%d\n", gen, batch, "CLRP", s_logical->id, t_logical->id, t_phisical->cur_t_dev_t);
#endif
						}
						else if (t_logical->working_batch != batch){
							switch (layer_task->l->type) {
							case LINK_FORWARD:
								default_pull_forward << <t_logical->size / thread_num + 1, t_logical->size>thread_num ? thread_num : t_logical->size, 0, streams[tasks] >> > (s_phisical->dev_t[s_phisical->cur_s_dev_t], t_phisical->dev_t[t_phisical->cur_t_dev_t], s_logical->offset, t_logical->offset, layer_task->l->dev_t, s_logical->size, gen);
								break;
							case LINK_FULL:
								if (s_logical->size <= t_logical->size) {
									default_pull_full << <t_logical->size / thread_num + 1, t_logical->size>thread_num ? thread_num : t_logical->size, 0, streams[tasks] >> > (s_phisical->dev_t[s_phisical->cur_s_dev_t], t_phisical->dev_t[t_phisical->cur_t_dev_t], s_logical->offset, t_logical->offset, layer_task->l->dev_t, s_logical->size, gen);
								}
								else {
									default_push_full << <s_logical->size / thread_num + 1, s_logical->size>thread_num ? thread_num : s_logical->size, 0, streams[tasks] >> > (s_phisical->dev_t[s_phisical->cur_s_dev_t], t_phisical->dev_t[t_phisical->cur_t_dev_t], s_logical->offset, t_logical->offset, layer_task->l->dev_t, t_logical->size, gen);
								}
							}
							tasks++;
							remove_executable(&layer_task_head, &layer_task_tail, layer_task);
							layer_task->done = true;
							t_logical->working_batch = batch;
#ifdef DEBUG
							fprintf(stdout, "GEN:%d BATCH:%d JOB:%s FROM:%d TO:%d BUFF:%d\n", gen, batch, "PUSH", s_logical->id, t_logical->id, t_phisical->cur_t_dev_t);
#endif
						}
						if (layer_task->l->mutated_gen != gen) {
							executable* n = new_executable(gen + 1, EXECUTE_LINK, s_logical, layer_task->l, layer_task->l->layer);
							append_executable(&link_task_head, &link_task_tail, n);
							layer_task->l->mutated_gen = gen;
						}
					}
				}
			}
			else {
				break;
			}
			executable* tmp = layer_task;
			layer_task = layer_task->next;
			if (tmp->done == true) {
				free(tmp);
			}
		}
		for (int i = 0; i < tasks; i++) {
			hipStreamSynchronize(streams[i]);
		}
		//critical region end
	}
}

void emit_layer(layer_t* l, float* t) {
	hipMemcpy(l->dev_t, t, l->size * sizeof(gen_t), hipMemcpyHostToDevice);
}

int main(int argc, char* argv[])
{
	/*
	int myid, numproces;
	int namelen;
	char processor_name[MPI_MAX_PROCESSOR_NAME];

	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &myid);
	MPI_Comm_size(MPI_COMM_WORLD, &numproces);
	MPI_Get_processor_name(processor_name, &namelen);
	char rbuf[10];
	if (myid == 4) {
		MPI_Recv(rbuf, 0, MPI_CHAR, 1, 0, MPI_COMM_WORLD, new MPI_Status());
		printf(rbuf);
	}
	if (myid == 1) {
		MPI_Send("hello send", 0, MPI_CHAR, 4, 0, MPI_COMM_WORLD);
	}
	Sleep(10000);
	MPI_Barrier(MPI_COMM_WORLD);
	*/

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		ERROR("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	thread_num = properties.maxThreadsPerBlock;

	w_mutes.size = 10;
	float host_t[10] = { 1.0 };
	hipMemcpyToSymbol(HIP_SYMBOL(w_mutes.dev_t), &host_t, w_mutes.size * sizeof(gen_w));

	has_layer_phsical(0, 9);

	has_layer_phsical(1, 9);
	has_layer_logical(11, 1, 0, 3, false);
	has_layer_logical(12, 1, 3, 3, false);
	has_layer_logical(13, 1, 6, 3, false);

	has_layer_phsical(21, 3);
	has_layer_logical(210, 21, 0, 3, true);
	has_layer_phsical(22, 3);
	has_layer_logical(220, 22, 0, 3, true);
	has_layer_phsical(23, 3);
	has_layer_logical(230, 23, 0, 3, true);

	has_layer_phsical(3, 3);
	has_layer_logical(30, 3, 0, 3, true);
	has_layer_logical(31, 3, 0, 1, false);
	has_layer_logical(32, 3, 3, 1, false);
	has_layer_logical(33, 3, 6, 1, false);

	has_link(0, LINK_FORWARD, NULL, 0, NULL, 1);

	has_link(1, LINK_FORWARD, NULL, 11, NULL, 21);
	has_link(2, LINK_FORWARD, NULL, 12, NULL, 22);
	has_link(3, LINK_FORWARD, NULL, 13, NULL, 23);

	has_link(4, LINK_FULL, NULL, 21, NULL, 31);
	has_link(5, LINK_FULL, NULL, 22, NULL, 32);
	has_link(6, LINK_FULL, NULL, 23, NULL, 33);

	has_link(7, LINK_FULL, NULL, 21, NULL, 210);
	has_link(8, LINK_FULL, NULL, 22, NULL, 220);
	has_link(9, LINK_FULL, NULL, 23, NULL, 230);

	has_link(10, LINK_FULL, NULL, 3, NULL, 30);

	default_init_device();

	friedrich_acts(net_events::EVENT_TEST, acts_test);

	friedrich_talking(9999);

	float input = 1.0;
	emit_layer(pick_layer(0), &input);

	execute(new_executable(1, EXECUTE_LAYER, pick_layer(0), pick_layer(0)->next, pick_layer(0)->next->layer),200);

	// Copy output vector from GPU buffer to host memory.
	//cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess) {
	//	ERROR("hipMemcpy failed!");
	//	goto Error;
	//}

	//printf("hello world! process %d of %d on %s\n", myid, numproces, processor_name);
	//MPI_Finalize();
	//system("pause");

Error:
	/*
	layer_t* iter = head;
	while (iter) {
		hipFree(iter->dev_t);
		link* iter2 = iter->next;
		while (iter2) {
			hipFree(iter2->dev_t);
			iter2 = iter2->another;
		}
		iter2 = iter->pre;
		while (iter2) {
			hipFree(iter2->dev_t);
			iter2 = iter2->another;
		}
	}
	*/
	system("pause");
    return 0;
}

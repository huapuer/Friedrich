#include "hip/hip_runtime.h"
/*
TODO: ������������ʼֵ
TODO: HEBBȨ����ǿ

DESG: �����߼��㣬�߼�������������һ���ֻ�ȫ������
	  �������������Լ����߼�������ã�����㱻����ʱ����ͬ�������߼���״̬��������߼��������������񵽵�����
DESG: Host Scheduler��Slave Batch���н��ʵ��Host��Slave������ҵ������Host�˵��ȿ���

TODO: ����������������߼���֮���״̬ͬ���߼�(scheduling debug), ������ɣ��������߼�������������(layer1->layer2)
TODO: ���Ӳ�ͬ���ӷ�ʽ��1:1/n:n��
TODO: ���Ӳ�����Ȩ������֧�֣�mute_fnΪNULL��
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "mpi.h" 
#include <Windows.h>
#include <memory.h>

#define ERROR(format,...) do{fprintf(stderr,format,##__VA_ARGS__);system("pause");exit(1);}while(0)
#define DEBUG

enum layer_type {
	LAYER_PHSICAL,
	LAYER_LOGICAL
};

enum execute_type {
	EXECUTE_LAYER,
	EXECUTE_LINK
};

enum layer_state {
	LAYER_STATE_INACTIVE,
	LAYER_STATE_INTEGRATE_ARRANGED,
	LAYER_STATE_INTEGRATING,
	LAYER_STATE_INTEGRATED,
	LAYER_STATE_EMITED
};

enum link_state {
	LINK_STATE_MUTATING,
	LINK_STATE_MUTATED
};

struct gen_t {
	long long gen;
	float t;
};

struct gen_w {
	long long gen;
	long long working_gen;
	int stage;
	float t;
};

struct link;

typedef void(*fp_integrate)(gen_t*, int);
typedef void(*fp_mute)(gen_w*, const long long);
typedef void(*fp_clear_push)(const gen_t *, gen_t *, int, gen_w*, const int, const long long);
typedef void(*fp_push)(const gen_t*, gen_t*, int, gen_w*, const int, const long long);

struct layer_t {
	int id;
	layer_state state;
	layer_type type;
	int size;
	link* pre;
	link* next;
	layer_t* follow;
	fp_integrate integrate_fn;
	fp_clear_push clear_push_fn;
	fp_push push_fn;

	//phsical
	gen_t *t;
	gen_t *dev_t;
	const float* dev_atte;
	layer_t* logical_head;
	layer_t* logical_tail;

	//logical
	int offset;
	bool delegate;
	layer_t* phsical;
	layer_t* next_logical;
};

struct link {
	long long gen;
	link_state state;
	layer_t* layer;
	int size;
	gen_w* t;
	gen_w *dev_t;
	link* another;
};

struct executable {
	long long gen;
	execute_type type;
	layer_t* s;
	link* l;
	layer_t* t;
	executable* pre;
	executable* next;
};

struct map {
	int size;
	float* dev_t;
};

__constant__ map w_mutes;
int thread_num;
layer_t* layer_list = 0;

__global__ void default_integrate(gen_t* s, int offset) {
	int i = threadIdx.x + offset;
	if (s[i].t > 3.0) {
		s[i].t = 1.0;
	}else {
		s[i].t = 0.0;
	}
}

__global__ void default_mute(gen_w* w, const long long gen) {
	int i = threadIdx.x;
	if (w[i].working_gen == gen) {
		int gap = gen - w[i].gen;
		if (gap == 1) {
			w[i].stage++;
		}
		else {
			w[i].stage = 0;
		}
		if (w[i].stage < w_mutes.size) {
			w[i].t += w_mutes.dev_t[w->stage];
		}
		w[i].gen = gen;
	}
}

__global__ void default_clear_push(const gen_t *s, gen_t *t, int to, gen_w* w, const int ss, const long long gen)
{
	int i = threadIdx.x + to;
	for (int j = 0; j < ss; j++) {
		t[i].t = 0.0;
		if (s[j].t > 0.0) {
			t[i].t += s[j].t * w[i*j + j].t;
			w[i*j + j].working_gen = gen;
		}
	}
}

__global__ void default_push(const gen_t *s, gen_t *t, int to, gen_w* w, const int ss, const long long gen)
{
	int i = threadIdx.x + to;
	for (int j = 0; j < ss; j++) {
		if (s[j].t > 0.0) {
			t[i].t += s[j].t * w[i*j + j].t;
			w[i*j + j].working_gen = gen;
		}
	}
}

layer_t* pick_layer(int idx) {
	if (!layer_list) {
		ERROR("COMPILE ERROR: LAYER[%d] NOT EXSISTS!\n", idx);
	}
	else {
		if (layer_list->id == idx) {
			return layer_list;
		}
		else {
			layer_t* iter = layer_list;
			while (iter->follow) {
				if (iter->follow->id == idx) {
					return iter->follow;
				}
				iter = iter->follow;
			}
		}
	}
	ERROR("COMPILE ERROR: LAYER[%d] NOT EXSISTS!\n", idx);
}

layer_t* new_layer_phsical(int id, int size,float atte=0.0, fp_integrate inte_fn=default_integrate, fp_clear_push cl_p_fn=default_clear_push, fp_push p_fn=default_push) {
	layer_t* ret = (layer_t*)malloc(sizeof(layer_t));
	memset(ret, 0, sizeof(layer_t));
	ret->id = id;
	ret->type = LAYER_PHSICAL;
	ret->size = size;
	ret->state = LAYER_STATE_INACTIVE;
	ret->offset = 0;
	ret->integrate_fn = inte_fn;
	ret->clear_push_fn = cl_p_fn;
	ret->push_fn = p_fn;

	if (atte > 0.0) {
		hipMemcpyToSymbol(HIP_SYMBOL(ret->dev_atte), &atte, sizeof(int));
	}
	if (size > 0) {
		ret->t = (gen_t*)malloc(sizeof(gen_t)*size);
		//TODO: initialize gen_t?
		hipMalloc((void**)&ret->dev_t, size * sizeof(gen_t));
		hipMemcpy(ret->dev_t, ret->t, size * sizeof(gen_t), hipMemcpyHostToDevice);
	}
	if (!layer_list) {
		layer_list = ret;
	}
	else {
		layer_t* iter = layer_list;
		while (iter->follow) {
			iter = iter->follow;
		}
		iter->follow = ret;
	}
	return ret;
}

layer_t* new_layer_logical(int id, int phsical, int offset, int size, bool delegate) {
	layer_t* ret = (layer_t*)malloc(sizeof(layer_t));
	memset(ret, 0, sizeof(layer_t));
	ret->id = id;
	ret->type = LAYER_LOGICAL;
	ret->size = size;
	ret->state = LAYER_STATE_INACTIVE;
	ret->delegate = delegate;

	layer_t* pl= pick_layer(phsical);
	ret->phsical = pl;
	ret->offset = offset;
	ret->t = pl->t;
	ret->dev_t = pl->dev_t;
	ret->integrate_fn = pl->integrate_fn;
	ret->clear_push_fn = pl->clear_push_fn;
	ret->push_fn = pl->push_fn;

	if (!pl->logical_head) {
		pl->logical_head = ret;
	}
	if (pl->logical_tail) {
		pl->logical_tail->next_logical = ret;
		pl->logical_tail = pl->logical_tail->next_logical;
	}
	else {
		pl->logical_tail = ret;
	}

	if (!layer_list) {
		layer_list = ret;
	}
	else {
		layer_t* iter = layer_list;
		while (iter->follow) {
			iter = iter->follow;
		}
		iter->follow = ret;
	}
	return ret;
}

link* new_link(layer_t* layer, int size) {
	link* ret = (link*)malloc(sizeof(link));
	memset(ret, 0, sizeof(link));
	ret->state = LINK_STATE_MUTATED;
	ret->layer = layer;
	ret->size = size;
	ret->t = (gen_w*)malloc(sizeof(gen_w)*size);
	//TODO: initialize gen_t
	hipMalloc((void**)&ret->dev_t, size * sizeof(gen_w));
	hipMemcpy(ret->dev_t, ret->t, size * sizeof(gen_w), hipMemcpyHostToDevice);
	return ret;
}

void add_link(link** head, link* next) {
	if(!*head) {
		*head = next;
		return;
	}
	else {
		link* tail = *head;
		while (tail->another) {
			tail = tail->another;
		}
		tail->another = next;
	}
}

layer_t* has_t(layer_t* s, int or_another_s, layer_t* next, int or_another_next) {
	if (!s) {
		s = pick_layer(or_another_s);
	}
	if (!s) {
		ERROR("COMPILE ERROR: LAYER[%d] NOT EXSISTS!\n", or_another_s);
	}

	if (!next) {
		next = pick_layer(or_another_next);
	}
	if (!next) {
		ERROR("COMPILE ERROR: LAYER[%d] NOT EXSISTS!\n", or_another_next);
	}
	int size = s->size*next->size;
	link* l = new_link(next, size);
	add_link(&s->next, l);
	//add_link(&next->pre, l);
	return next;
}

executable* new_executable(int gen, execute_type type, layer_t* s, link* l, layer_t* t) {
	executable* ret = (executable*)malloc(sizeof(executable));
	memset(ret, 0, sizeof(executable));
	ret->gen = gen;
	ret->type = type;
	ret->s = s;
	ret->l = l;
	ret->t = t;
	return ret;
}

fp_mute mute_fn;

void append_executable(executable** head, executable** tail, executable* n) {
	if (!*head) {
		*head = n;
	}
	if (*tail) {
		(*tail)->next = n;
		(*tail)->next->pre = *tail;
		(*tail) = (*tail)->next;
	}
	else {
		*tail = n;
	}
}

void remove_executable(executable** head, executable** tail, executable* e) {
	if (e->pre) {
		e->pre->next = e->next;
		if (!e->pre->pre) {
			*head = e->pre;
			if (*head) {
				(*head)->next = e->next;
			}
		}
	}
	else {
		*head = e->next;
		if (*head) {
			(*head)->next = e->next->next;
		}
	}
	if (e->next) {
		e->next->pre = e->pre;
		if (!e->next->next) {
			*tail = e->next;
			if (*tail) {
				(*tail)->pre = e->pre;
			}
		}
	}
	else {
		*tail = e->pre;
		if (*tail) {
			(*tail)->pre = e->pre->pre;
		}
	}
}

void execute(executable* head, int max_gen) {
	long long gen = 1;
	long long batch = 0;
	const int task_width = 10;
	int tasks = 0;
	hipStream_t streams[task_width];
	for (int i = 0; i<task_width; i++)
	{
		hipStreamCreate(&streams[i]);
	}

	executable* link_task_head = NULL;
	executable* link_task_tail = NULL;
	executable* layer_task_head = head;
	executable* layer_task_tail = head;
	executable* inqueue_task_head = NULL;
	executable* inqueue_task_tail = NULL;

	while (layer_task_head) {
		//critical region begin
		//excute
		
		batch++;
		tasks = 0;

		if (layer_task_head->gen > gen) {
#ifdef DEBUG
			if (gen >= max_gen) {
				break;
			}
#endif
			gen++;

			executable* link_task = link_task_head;
			while (link_task) {			
				while (link_task && tasks < task_width) {
					mute_fn << <link_task->l->size / thread_num + 1, link_task->l->size>thread_num ? thread_num : link_task->l->size, 0, streams[tasks] >> >(link_task->l->dev_t, gen);
					remove_executable(&link_task_head, &link_task_tail, link_task);
					append_executable(&inqueue_task_head, &inqueue_task_tail, link_task);
					tasks++;
					link_task->l->state = LINK_STATE_MUTATING;
#ifdef DEBUG
					fprintf(stdout, "GEN:%d BATCH:%d JOB:%s FROM:%d TO:%d\n", gen, batch, "MUTW", link_task->s->id, link_task->t->id);
#endif
					link_task = link_task->next;
				}
				if (tasks == task_width) {
					for (int i = 0; i < tasks; i++) {
						hipStreamSynchronize(streams[i]);
					}
					batch++;
					tasks = 0;
				}
			}

			pick_layer(0)->state = LAYER_STATE_INTEGRATED;
			layer_task_head->pre=new_executable(gen, EXECUTE_LAYER, pick_layer(0), pick_layer(0)->next, pick_layer(0)->next->layer);
			layer_task_head->pre->next = layer_task_head;
			layer_task_head = layer_task_head->pre;
		}

		executable* layer_task = layer_task_head;
		while (layer_task && tasks < task_width) {
			if (layer_task->gen == gen && tasks < task_width) {
				if (layer_task->s->state == LAYER_STATE_INTEGRATING) {
					layer_task->s->integrate_fn << <layer_task->s->size / thread_num + 1, layer_task->s->size>thread_num ? thread_num : layer_task->s->size, 0, streams[tasks] >> >(layer_task->s->dev_t, layer_task->s->offset);
					append_executable(&inqueue_task_head, &inqueue_task_tail, layer_task);
					layer_task->s->state = LAYER_STATE_INTEGRATE_ARRANGED;
#ifdef DEBUG
					fprintf(stdout, "GEN:%d BATCH:%d JOB:%s FROM:%d TO:%d\n", gen, batch, "INTE", layer_task->s->id, layer_task->t->id);
#endif
				}
				else {
					if (layer_task->s->state == LAYER_STATE_INTEGRATED && layer_task->l->state == LINK_STATE_MUTATED && layer_task->t->state != LAYER_STATE_INTEGRATE_ARRANGED && layer_task->t->state != LAYER_STATE_INTEGRATED) {
						if (layer_task->t->state != LAYER_STATE_INTEGRATING) {
							layer_task->s->clear_push_fn << <layer_task->t->size / thread_num + 1, layer_task->t->size>thread_num ? thread_num : layer_task->t->size, 0, streams[tasks] >> >(layer_task->s->dev_t, layer_task->t->dev_t, layer_task->t->offset, layer_task->l->dev_t, layer_task->s->size, gen);
							remove_executable(&layer_task_head, &layer_task_tail, layer_task);
							append_executable(&inqueue_task_head, &inqueue_task_tail, layer_task);
							layer_task->t->state = LAYER_STATE_INTEGRATING;
							link* next_link = layer_task->t->next;
							while (next_link) {
								executable* n = new_executable(gen + 1, EXECUTE_LAYER, layer_task->t, next_link, next_link->layer);
								append_executable(&layer_task_head, &layer_task_tail, n);
								next_link = next_link->another;
							}
							switch (layer_task->t->type) {
							case LAYER_LOGICAL:
								if (!layer_task->t->delegate) {
									link* next = layer_task->t->phsical->next;
									while (next) {
										executable* n = new_executable(gen + 1, EXECUTE_LAYER, layer_task->t->phsical, next, next->layer);
										append_executable(&layer_task_head, &layer_task_tail, n);
										next = next->another;
									}
								}
								layer_task->t->phsical->state = LAYER_STATE_INTEGRATING;
								break;
							case LAYER_PHSICAL:
								layer_t* next = layer_task->t->logical_head;
								while (next) {
									if (!next->delegate) {
										link* next_link = next->next;
										while (next_link) {
											executable* n = new_executable(gen + 1, EXECUTE_LAYER, next, next_link, next_link->layer);
											append_executable(&layer_task_head, &layer_task_tail, n);
											next_link = next_link->another;
										}
									}
									next->state = LAYER_STATE_INTEGRATING;
									next = next->next_logical;
								}
								break;
							}
#ifdef DEBUG
							fprintf(stdout, "GEN:%d BATCH:%d JOB:%s FROM:%d TO:%d\n", gen, batch, "CLRP", layer_task->s->id, layer_task->t->id);
#endif
						}
						else {
							layer_task->s->push_fn << <layer_task->t->size / thread_num + 1, layer_task->t->size>thread_num ? thread_num : layer_task->t->size, 0, streams[tasks] >> > (layer_task->s->dev_t, layer_task->t->dev_t, layer_task->t->offset, layer_task->l->dev_t, layer_task->s->size, gen);
							remove_executable(&layer_task_head, &layer_task_tail, layer_task);
							append_executable(&inqueue_task_head, &inqueue_task_tail, layer_task);
#ifdef DEBUG
							fprintf(stdout, "GEN:%d BATCH:%d JOB:%s FROM:%d TO:%d\n", gen, batch, "PUSH", layer_task->s->id, layer_task->t->id);
#endif
						}
						executable* n = new_executable(gen + 1, EXECUTE_LINK, layer_task->s, layer_task->l, layer_task->l->layer);
						append_executable(&link_task_head, &link_task_tail, n);
					}
				}
				tasks++;
			}
			else {
				break;
			}
			layer_task = layer_task->next;
		}
		for (int i = 0; i < tasks; i++) {
			hipStreamSynchronize(streams[i]);
		}
		executable* inqueue_task = inqueue_task_head;
		while (inqueue_task) {
			executable* next_inqueue_task = inqueue_task->next;
			switch (inqueue_task->type) {
			case EXECUTE_LINK:
				inqueue_task->l->state = LINK_STATE_MUTATED;
				remove_executable(&inqueue_task_head, &inqueue_task_tail, inqueue_task);
				free(inqueue_task);
				break;
			case EXECUTE_LAYER:
				layer_state update;
				layer_t* s = inqueue_task->s;
				switch (s->state) {
				case LAYER_STATE_INTEGRATE_ARRANGED:
					update = LAYER_STATE_INTEGRATED;
					remove_executable(&inqueue_task_head, &inqueue_task_tail, inqueue_task);
					break;
				case LAYER_STATE_INTEGRATED:
					update = LAYER_STATE_EMITED;
					remove_executable(&inqueue_task_head, &inqueue_task_tail, inqueue_task);
					free(inqueue_task);
					break;
				}
				switch (s->type) {
				case LAYER_LOGICAL:
					s->phsical->state = update;
					break;
				case LAYER_PHSICAL:
					layer_t* next = s->logical_head;
					while (next) {
						next->state = update;
						next = next->next_logical;
					}
					break;
				}
				s->state = update;
				break;
			}
			inqueue_task = next_inqueue_task;
		}
		//critical region end
	}
}

void emmit_layer(layer_t* l, float* t) {
	hipMemcpy(l->dev_t, t, l->size * sizeof(gen_t), hipMemcpyHostToDevice);
	l->state = LAYER_STATE_INTEGRATED;
}

int main(int argc, char* argv[])
{
	/*
	int myid, numproces;
	int namelen;
	char processor_name[MPI_MAX_PROCESSOR_NAME];

	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &myid);
	MPI_Comm_size(MPI_COMM_WORLD, &numproces);
	MPI_Get_processor_name(processor_name, &namelen);
	char rbuf[10];
	if (myid == 4) {
		MPI_Recv(rbuf, 0, MPI_CHAR, 1, 0, MPI_COMM_WORLD, new MPI_Status());
		printf(rbuf);
	}
	if (myid == 1) {
		MPI_Send("hello send", 0, MPI_CHAR, 4, 0, MPI_COMM_WORLD);
	}
	Sleep(10000);
	MPI_Barrier(MPI_COMM_WORLD);
	*/

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		ERROR("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	thread_num = properties.maxThreadsPerBlock;

	w_mutes.size = 10;
	float host_t[10] = { 1.0 };
	hipMemcpyToSymbol(HIP_SYMBOL(w_mutes.dev_t), &host_t, w_mutes.size * sizeof(gen_w));

	mute_fn = default_mute;

	new_layer_phsical(0, 9);

	new_layer_phsical(1, 9);
	new_layer_logical(11, 1, 0, 3, false);
	new_layer_logical(12, 1, 3, 3, false);
	new_layer_logical(13, 1, 6, 3, false);

	new_layer_phsical(21, 3);
	new_layer_logical(210, 21, 0, 3, true);
	new_layer_phsical(22, 3);
	new_layer_logical(220, 22, 0, 3, true);
	new_layer_phsical(23, 3);
	new_layer_logical(230, 23, 0, 3, true);

	new_layer_phsical(3, 3);
	new_layer_logical(30, 3, 0, 3, true);
	new_layer_logical(31, 3, 0, 1, false);
	new_layer_logical(32, 3, 3, 1, false);
	new_layer_logical(33, 3, 6, 1, false);

	has_t(NULL, 0, NULL, 1);

	has_t(NULL, 11, NULL, 21);
	has_t(NULL, 12, NULL, 22);
	has_t(NULL, 13, NULL, 23);

	has_t(NULL, 21, NULL, 31);
	has_t(NULL, 210, NULL, 31);
	has_t(NULL, 22, NULL, 32);
	has_t(NULL, 220, NULL, 32);
	has_t(NULL, 23, NULL, 33);
	has_t(NULL, 230, NULL, 33);

	has_t(NULL, 21, NULL, 210);
	has_t(NULL, 210, NULL, 21);
	has_t(NULL, 22, NULL, 220);
	has_t(NULL, 220, NULL, 22);
	has_t(NULL, 23, NULL, 230);
	has_t(NULL, 230, NULL, 23);

	has_t(NULL, 3, NULL, 30);
	has_t(NULL, 30, NULL, 3);

	float input = 1.0;
	emmit_layer(pick_layer(0), &input);

	execute(new_executable(1, EXECUTE_LAYER, pick_layer(0), pick_layer(0)->next, pick_layer(0)->next->layer),200);

	// Copy output vector from GPU buffer to host memory.
	//cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess) {
	//	ERROR("hipMemcpy failed!");
	//	goto Error;
	//}

	//printf("hello world! process %d of %d on %s\n", myid, numproces, processor_name);
	//MPI_Finalize();
	//system("pause");

Error:
	/*
	layer_t* iter = head;
	while (iter) {
		hipFree(iter->dev_t);
		link* iter2 = iter->next;
		while (iter2) {
			hipFree(iter2->dev_t);
			iter2 = iter2->another;
		}
		iter2 = iter->pre;
		while (iter2) {
			hipFree(iter2->dev_t);
			iter2 = iter2->another;
		}
	}
	*/
	system("pause");
    return 0;
}

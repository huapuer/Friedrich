#include "hip/hip_runtime.h"
/*
TODO: XORʱ������
TODO: ��ӡDEBUG��Ϣ
TODO: ������������ʼֵ
TODO: HEBBȨ����ǿ

DESG: �����߼��㣬�߼�������������һ���ֻ�ȫ������
	  �������������Լ����߼�������ã�����㱻����ʱ����ͬ�������߼���״̬��������߼��������������񵽵�����
DESG: Host Scheduler��Slave Batch���н��ʵ��Host��Slave������ҵ������Host�˵��ȿ���
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "mpi.h" 
#include <Windows.h>
#include <memory.h>

#define error(stream,format,...) do{fprintf(stream,format,##__VA_ARGS__);system("pause");exit(1);}while(0)
#define DEBUG

enum layer_type {
	LAYER_I,
	LAYER_C,
	LAYER_XOR,
	LAYER_V
};

enum execute_type {
	EXECUTE_LAYER,
	EXECUTE_LINK,
	EXECUTE_INTEGRATE
};

struct gen_t {
	long long gen;
	float t;
};

struct gen_w {
	long long gen;
	long long working_gen;
	int stage;
	float t;
};

struct link;

struct layer_t {
	long long gen;
	int id;
	layer_type type;
	int size;
	int integrating_batch;
	int working_batch;
	gen_t *t;
	gen_t *dev_t;
	const float* dev_atte;
	link* pre;
	link* next;
	layer_t* follow;
};

struct link {
	long long gen;
	layer_t* layer;
	int size;
	gen_w* t;
	gen_w *dev_t;
	link* another;
};

struct executable {
	long long gen;
	execute_type type;
	layer_t* s;
	link* l;
	layer_t* t;
	executable* pre;
	executable* next;
};

struct map {
	int size;
	float* dev_t;
};

__constant__ map w_mutes;
int thread_num;
layer_t* list = 0;
layer_t* head = 0;

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void inte_i(gen_t* s) {
	int i = threadIdx.x;
	if (s[i].t > 3.0) {
		s[i].t = 1.0;
	}else {
		s[i].t = 0.0;
	}
}

__global__ void inte_c(gen_t* s) {
	int i = threadIdx.x;
	if (s[i].t > 3.0) {
		s[i].t = 1.0;
	}else {
		s[i].t = 0.0;
	}
}

__global__ void inte_xor(gen_t* s) {
	int i = threadIdx.x;
	if (s[i].t == 2.0) {
		s[i].t = 1.0;
	}else {
		s[i].t = 0.0;
	}
}

__global__ void mute_w(gen_w* w, const long long gen) {
	int i = threadIdx.x;
	if (w[i].working_gen == gen) {
		int gap = gen - w[i].gen;
		if (gap == 1) {
			w[i].stage++;
		}
		else {
			w[i].stage = 0;
		}
		if (w[i].stage < w_mutes.size) {
			w[i].t += w_mutes.dev_t[w->stage];
		}
		w[i].gen = gen;
	}
}

__global__ void i_2_atte_i(const gen_t *s, gen_t *t, gen_w* w, const int ss, const long long gen)
{
	int i = threadIdx.x;
	for (int j = 0; j < ss; j++) {
		t[i].t = 0.0;
		if (s[j].t > 0.0) {
			t[i].t += s[j].t * w[i*j + j].t;
			w[i*j + j].working_gen = gen;
		}
	}
}

__global__ void i_2_i(const gen_t *s, gen_t *t, gen_w* w, const int ss, const long long gen)
{
	int i = threadIdx.x;
	for (int j = 0; j < ss; j++) {
		if (s[j].t > 0.0) {
			t[i].t += s[j].t * w[i*j + j].t;
			w[i*j + j].working_gen = gen;
		}
	}
}

__global__ void i_2_c(const gen_t *s, gen_t *t, const float* atte, const long long gen)
{
	int i = threadIdx.x;
	int gap = gen - t[i].gen;
	if (gap > 10) {
		t[i].t = 0.0;
	}
	else {
		t[i].t = t[i].t * pow((double)*atte, (double)gap);
	}
	t[i].t += s[i].t;
}

__global__ void i_2_atte_xor(const gen_t *s, gen_t *t)
{
	int i = threadIdx.x;
	t[i].t = 0.0;
	t[i].t += s[i].t > 0 ? 1.0 : 0.0;
}

__global__ void i_2_xor(const gen_t *s, gen_t *t)
{
	int i = threadIdx.x;
	t[i].t += s[i].t > 0 ? 1.0 : 0.0;
}

__global__ void c_2_atte_i(const gen_t *s, gen_t *t)
{
	int i = threadIdx.x;
	t[i].t = 0.0;
	t[i].t -= s[i].t;
}

__global__ void c_2_i(const gen_t *s, gen_t *t)
{
	int i = threadIdx.x;
	t[i].t -= s[i].t;
}

__global__ void xor_2_atte_i(const gen_t *s, gen_t *t)
{
	int i = threadIdx.x;
	t[i].t = 0.0;
	t[i].t += s[i].t;
}

__global__ void xor_2_i(const gen_t *s, gen_t *t)
{
	int i = threadIdx.x;
	t[i].t += s[i].t;
}

__global__ void v_2_atte_xor(const gen_t *s, gen_t *t)
{
	int i = threadIdx.x;
	t[i].t = 0.0;
	t[i].t += s[i].t > 0 ? 1.0 : 0.0;
	//printf("DEBUG v_2_atte_xor %d: f=%f\n", threadIdx.x, t[i].t);
}

__global__ void v_2_xor(const gen_t *s, gen_t *t)
{
	int i = threadIdx.x;
	t[i].t += s[i].t > 0 ? 1.0 : 0.0;
}

layer_t* new_layer(int id, layer_type type, int size,float atte=0.0) {
	layer_t* ret = (layer_t*)malloc(sizeof(layer_t));
	memset(ret, 0, sizeof(layer_t));
	ret->id = id;
	ret->type = type;
	ret->size = size;
	ret->gen = 0;
	ret->working_batch = 0;
	if (atte > 0.0) {
		hipMemcpyToSymbol(HIP_SYMBOL(ret->dev_atte), &atte, sizeof(int));
	}
	if (size > 0) {
		ret->t = (gen_t*)malloc(sizeof(gen_t)*size);
		//TODO: initialize gen_t?
		hipMalloc((void**)&ret->dev_t, size * sizeof(gen_t));
		hipMemcpy(ret->dev_t, ret->t, size * sizeof(gen_t), hipMemcpyHostToDevice);
	}
	if (!list) {
		list = ret;
	}
	else {
		layer_t* iter = list;
		while (iter->follow) {
			iter = iter->follow;
		}
		iter->follow = ret;
	}
	return ret;
}

link* new_link(layer_t* layer, int size) {
	link* ret = (link*)malloc(sizeof(link));
	memset(ret, 0, sizeof(link));
	ret->layer = layer;
	ret->size = size;
	ret->t = (gen_w*)malloc(sizeof(gen_w)*size);
	//TODO: initialize gen_t
	hipMalloc((void**)&ret->dev_t, size * sizeof(gen_w));
	hipMemcpy(ret->dev_t, ret->t, size * sizeof(gen_w), hipMemcpyHostToDevice);
	return ret;
}

layer_t* pick_layer(int idx) {
	if (!list) {
		error(stderr, "COMPILE ERROR: LAYER[%d] NOT EXSISTS!\n", idx);
	}
	else {
		if (list->id == idx) {
			return list;
		}
		else {
			layer_t* iter = list;
			while (iter->follow) {
				if (iter->follow->id == idx) {
					return iter->follow;
				}
				iter = iter->follow;
			}
		}
	}
	error(stderr, "COMPILE ERROR: LAYER[%d] NOT EXSISTS!\n", idx);
}

void add_link(link** head, link* next) {
	if(!*head) {
		*head = next;
		return;
	}
	else {
		link* tail = *head;
		while (tail->another) {
			tail = tail->another;
		}
		tail->another = next;
	}
}

layer_t* has_t(layer_t* s, int or_another_s, layer_t* next, int or_another_next) {
	if (!head) {
		if (s) {
			head = s;
		}
		else {
			error(stderr, "COMPILE ERROR: HEAD OF LAYER NOT EXSISTS!\n");
		}
	}
	if (or_another_s) {
		s = pick_layer(or_another_s);
	}
	if (!s) {
		error(stderr, "COMPILE ERROR: LAYER[%d] NOT EXSISTS!\n", or_another_s);
	}
	if (or_another_next) {
		next = pick_layer(or_another_next);
	}
	if (!next) {
		error(stderr, "COMPILE ERROR: LAYER[%d] NOT EXSISTS!\n", or_another_next);
	}
	int size = 0;
	switch (s->type) {
	case LAYER_I:
		switch (next->type) {
		case LAYER_I:
			size = s->size*next->size;
			break;
		case LAYER_C:
			if (s->size != next->size) {
				error(stderr, "COMPILE ERROR: LAYER[%d] SIZE UNMATCHED!\n", next->id);
			}
			size = 0;
			break;
		case LAYER_XOR:
			if (next->pre && next->pre->another) {
				error(stderr, "COMPILE ERROR: XOR LAYER[%d] OUT OF INPUTS!\n", next->id);
			}
			size = 0;
			break;
		default:
			error(stderr, "COMPILE ERROR: LAYER[%d] TYPE UNMATCHED!\n", next->id);
		}
		break;
	case LAYER_C:
		switch (next->type) {
		case LAYER_I:
			if (s->size != next->size) {
				error(stderr, "COMPILE ERROR: LAYER[%d] SIZE UNMATCHED!\n", next->id);
			}
			size = 0;
			break;
		default:
			error(stderr, "COMPILE ERROR: LAYER[%d] TYPE UNMATCHED!\n", next->id);
		}
		break;
	case LAYER_XOR:
		switch (next->type) {
		case LAYER_I:
			size = s->size*next->size;
			break;
		default:
			error(stderr, "COMPILE ERROR: LAYER[%d] TYPE UNMATCHED!\n", next->id);
		}
		break;
	case LAYER_V:
		switch (next->type) {
		case LAYER_XOR:
			if (s->size != next->size) {
				error(stderr, "COMPILE ERROR: LAYER[%d] SIZE UNMATCHED!\n", next->id);
			}
			if (next->pre && next->pre->another) {
				error(stderr, "COMPILE ERROR: XOR LAYER[%d] OUT OF INPUTS!\n", next->id);
			}
			size = 0;
			break;
		default:
			error(stderr, "COMPILE ERROR: LAYER[%d] TYPE UNMATCHED!\n", next->id);
		}
		break;
	default:
		error(stderr, "COMPILE ERROR: LAYER_TYPE[%d] UNDEFINED!\n", s->type);
	}
	link* l = new_link(next, size);
	add_link(&s->next, l);
	//add_link(&next->pre, l);
	return next;
}

executable* new_executable(int gen, execute_type type, layer_t* s, link* l, layer_t* t) {
	executable* ret = (executable*)malloc(sizeof(executable));
	memset(ret, 0, sizeof(executable));
	ret->gen = gen;
	ret->type = type;
	ret->s = s;
	ret->l = l;
	ret->t = t;
	return ret;
}

int launch_job(executable** head, executable** tail, executable* e, int gen, hipStream_t stream) {
	if (e->type == EXECUTE_LINK) {
		mute_w <<<e->l->size / thread_num + 1, e->l->size>thread_num?thread_num:e->l->size, 0, stream >>> (e->l->dev_t, gen);
		return 1;
	}
	switch (e->s->type) {
	case LAYER_I:
		if (e->s->gen < gen) {
			inte_i <<<e->s->size / thread_num + 1, e->s->size>thread_num?thread_num:e->s->size, 0, stream>>> (e->s->dev_t);
			e->s->gen = gen;
			return 0;
		}
		else {
			switch (e->t->type) {
			case LAYER_I:
				if (e->t->gen < gen) {
					i_2_atte_i <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t, e->l->dev_t, e->s->size, gen);
					//e->t->gen = gen;
				}
				else {
					i_2_i <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t, e->l->dev_t, e->s->size, gen);
				}
				if (e->l->gen < gen) {
					executable* n = new_executable(gen + 1, EXECUTE_LINK, e->s, e->l, e->l->layer);
					/*
					if (head) {
						head->pre = n;
						head->pre->next = head;
						head = head->pre;
					}
					else {
						head = n;
					}
					*/
					if (*tail) {
						(*tail)->next = n;
						(*tail)->next->pre = *tail;
						(*tail) = (*tail)->next;
					}
					else {
						(*tail) = n;
					}
					e->l->gen = gen;
				}
				break;
			case LAYER_C:
				i_2_c <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>>(e->s->dev_t, e->t->dev_t, e->t->dev_atte, gen);
				break;
			case LAYER_XOR:
				if (e->t->gen < gen) {
					i_2_atte_xor <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t);
					//e->t->gen = gen;
				}
				else {
					i_2_xor <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t);
				}
				break;
			default:
				error(stderr, "EXECUTE ERROR: LAYER_TYPE[%d] UNDEFINED OR UNMATCHED!\n", e->t->type);
			}
			return 1;
		}
		break;
	case LAYER_C:
		if (e->s->gen < gen) {
			inte_c <<<e->s->size / thread_num + 1, e->s->size>thread_num?thread_num:e->s->size, 0, stream >>> (e->s->dev_t);
			e->s->gen = gen;
			return 0;
		}
		else {
			switch (e->t->type) {
			case LAYER_I:
				if (e->t->gen < gen) {
					c_2_atte_i <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t);
					//e->t->gen = gen;
				}
				else {
					c_2_i <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t);
				}
				break;
			default:
				error(stderr, "EXECUTE ERROR: LAYER_TYPE[%d] UNDEFINED OR UNMATCHED!\n", e->t->type);
			}
			return 1;
		}
		break;
	case LAYER_XOR:
		if (e->s->gen < gen) {
			inte_xor <<<e->s->size / thread_num + 1, e->s->size>thread_num?thread_num:e->s->size, 0, stream >>> (e->s->dev_t);
			e->s->gen = gen;
			return 0;
		}
		else {
			switch (e->t->type) {
			case LAYER_I:
				if (e->t->gen < gen) {
					xor_2_atte_i <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t);
					//e->t->gen = gen;
				}
				else {
					xor_2_i <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t);
				}
				break;
			default:
				error(stderr, "EXECUTE ERROR: LAYER_TYPE[%d] UNDEFINED OR UNMATCHED!\n", e->t->type);
			}
			return 1;
		}
		break;
	case LAYER_V:
		switch (e->t->type) {
		case LAYER_XOR:
			if (e->t->gen < gen) {
				v_2_atte_xor <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t);
				//e->t->gen = gen;
			}
			else {
				v_2_xor <<<e->t->size / thread_num + 1, e->t->size>thread_num?thread_num:e->t->size, 0, stream >>> (e->s->dev_t, e->t->dev_t);
			}
			return 1;
		default:
			error(stderr, "EXECUTE ERROR: LAYER_TYPE[%d] UNDEFINED OR UNMATCHED!\n", e->t->type);
		}
		break;
	default:
		error(stderr, "EXECUTE ERROR: LAYER_TYPE[%d] UNDEFINED!\n", e->s->type);
	}
}

void execute(executable* e, int max_gen) {
	long long gen = 1;
	long long batch = 0;
	const int task_width = 10;
	hipStream_t streams[task_width];
	for (int i = 0; i<task_width; i++)
	{
		hipStreamCreate(&streams[i]);
	}
	executable* head = e;
	executable* tail = e;
	while (head) {
		//critical region begin
		//excute
		if (head->gen > gen) {
#ifdef DEBUG
			if (gen >= max_gen) {
				break;
			}
#endif
			gen++;
		}
		batch++;
		int tasks = 0;
		e = head;
		while (e && tasks < task_width) {
			if (e->gen == gen && tasks < task_width) {
				if (e->type == EXECUTE_LINK || (e->type==EXECUTE_LAYER && e->s->integrating_batch < batch && e->t->working_batch < batch)) {
					if (launch_job(&head, &tail, e, gen, streams[tasks]) == 1) {
#ifdef DEBUG
						fprintf(stdout, "GEN:%d BATCH:%d JOB:%d FROM:%d TO:%d FROM_WORKIN:%d TO_WORKING:%d\n", gen, batch, e->type, e->s->id, e->t->id, e->s->working_batch, e->t->working_batch);
#endif
						e->t->working_batch = batch;
						if (e->pre) {
							e->pre->next = e->next;
							if (!e->pre->pre) {
								head = e->pre;
								if (head) {
									head->next = e->next;
								}
							}
						}
						else {
							head = e->next;
							if (head) {
								head->next = e->next->next;
							}
						}
						if (e->next) {
							e->next->pre = e->pre;
							if (!e->next->next) {
								tail = e->next;
								if (tail) {
									tail->pre = e->pre;
								}
							}
						}
						else {
							tail = e->pre;
							if (tail) {
								tail->pre = e->pre->pre;
							}
						}
						if (e->type == EXECUTE_LAYER && e->t->gen < gen) {
							link* next = e->t->next;
							while (next) {
								executable* n = new_executable(gen + 1, EXECUTE_LAYER, e->t, next, next->layer);
								if (!head) {
									head = n;
								}
								if (tail) {
									tail->next = n;
									tail->next->pre = tail;
									tail = tail->next;
								}
								else {
									tail = n;
								}
								next = next->another;
							}
							e->t->gen = gen;
						}
						executable* f = e;
						e = e->next;
						free(f);
						continue;
					}
					else {
						e->s->integrating_batch = batch;
#ifdef DEBUG
						fprintf(stdout, "GEN:%d BATCH:%d JOB:%d FROM:%d TO:%d\n", gen, batch, EXECUTE_INTEGRATE, e->s->id, e->t->id);
#endif
					}
					tasks++;
				}
			}
			else {
				break;
			}
			e = e->next;
		}
		for (int i = 0; i < tasks; i++) {
			hipStreamSynchronize(streams[i]);
		}
		//critical region end
	}
	for (int i = 0; i<5; i++)
	{
		hipStreamDestroy(streams[i]);
	}
}

void emmit_layer(layer_t* l, float* t) {
	hipMemcpy(l->dev_t, t, l->size * sizeof(gen_t), hipMemcpyHostToDevice);
}

int main(int argc, char* argv[])
{
	/*
	int myid, numproces;
	int namelen;
	char processor_name[MPI_MAX_PROCESSOR_NAME];

	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &myid);
	MPI_Comm_size(MPI_COMM_WORLD, &numproces);
	MPI_Get_processor_name(processor_name, &namelen);
	char rbuf[10];
	if (myid == 4) {
		MPI_Recv(rbuf, 0, MPI_CHAR, 1, 0, MPI_COMM_WORLD, new MPI_Status());
		printf(rbuf);
	}
	if (myid == 1) {
		MPI_Send("hello send", 0, MPI_CHAR, 4, 0, MPI_COMM_WORLD);
	}
	Sleep(10000);
	MPI_Barrier(MPI_COMM_WORLD);
	*/

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		error(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	thread_num = properties.maxThreadsPerBlock;

	w_mutes.size = 10;
	float host_t[10] = { 1.0 };
	hipMemcpyToSymbol(HIP_SYMBOL(w_mutes.dev_t), &host_t, w_mutes.size * sizeof(gen_w));

	has_t(new_layer(0, LAYER_V, 1), 0, new_layer(1, LAYER_XOR, 1), 0);
	has_t(NULL, 1, new_layer(2, LAYER_I, 2), 0);
	has_t(NULL, 2, new_layer(3, LAYER_I, 2), 0);
	has_t(NULL, 3, new_layer(4, LAYER_I, 2), 0);
	has_t(NULL, 4, new_layer(5, LAYER_I, 1), 0);
	has_t(NULL, 5, NULL, 1);
	has_t(NULL, 4, new_layer(6, LAYER_I, 2), 0);
	has_t(NULL, 3, new_layer(7, LAYER_C, 2), 0);
	has_t(NULL, 7, NULL, 4);

	float input = 1.0;
	emmit_layer(head, &input);

	execute(new_executable(1, EXECUTE_LAYER, head, head->next, head->next->layer),20);

	// Copy output vector from GPU buffer to host memory.
	//cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess) {
	//	error(stderr, "hipMemcpy failed!");
	//	goto Error;
	//}

	//printf("hello world! process %d of %d on %s\n", myid, numproces, processor_name);
	//MPI_Finalize();
	//system("pause");

Error:
	/*
	layer_t* iter = head;
	while (iter) {
		hipFree(iter->dev_t);
		link* iter2 = iter->next;
		while (iter2) {
			hipFree(iter2->dev_t);
			iter2 = iter2->another;
		}
		iter2 = iter->pre;
		while (iter2) {
			hipFree(iter2->dev_t);
			iter2 = iter2->another;
		}
	}
	*/
	system("pause");
    return 0;
}
